#include <assert.h>
#include <stdio.h>
#include "KernelAdd.cuh"

int main() {
  int numElements = 1 << 28;
	float *x = NULL;
  float *y = NULL;
  float *result = NULL;

	hipMallocManaged(&x, numElements * sizeof(*x));
	hipMallocManaged(&y, numElements * sizeof(*y));
	hipMallocManaged(&result, numElements * sizeof(*result));


	for (int i = 0; i < numElements; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blockSize = 256;

	int numBlocks = (numElements + blockSize - 1) / blockSize;

  hipEvent_t start;
  hipEvent_t stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
	KernelAdd<<<numBlocks, blockSize>>>(numElements, x, y, result);
  hipEventRecord(stop);

  float millis = 0;
  hipEventElapsedTime(&millis, start, stop);
  printf("Elpased: %f\n", millis);

	hipDeviceSynchronize();

  for(int i = 0; i < numElements; ++i) {
    assert(result[i] == 3.0f);
  }

  hipFree(x);
  hipFree(y);
  hipFree(result);

  return 0;
}
