#include <assert.h>
#include "KernelMul.cuh"

int main() {
  int numElements = 1 << 28;
	float *x = NULL;
  float *y = NULL;
  float *result = NULL;

	hipMallocManaged(&x, numElements * sizeof(*x));
	hipMallocManaged(&y, numElements * sizeof(*y));
	hipMallocManaged(&result, numElements * sizeof(*result));


	for (int i = 0; i < numElements; ++i) {
		x[i] = 3.0f;
		y[i] = 2.0f;
	}

	int blockSize = 256;

	int numBlocks = (numElements + blockSize - 1) / blockSize;

	KernelMul<<<numBlocks, blockSize>>>(numElements, x, y, result);
	hipDeviceSynchronize();

  for(int i = 0; i < numElements; ++i) {
    assert(result[i] == 6.0f);
  }

  hipFree(x);
  hipFree(y);
  hipFree(result);

  return 0;
}
