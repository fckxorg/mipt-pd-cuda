#include <assert.h>
#include "KernelMatrixAdd.cuh"

void FillMatrix(float* mat, int width, int height, float value) {
  for(int row = 0; row < height; ++row) {
    for(int col = 0; col < width; ++col) {
      mat[row * width + col] = value;
    }
  }
}

int main() {
  int width = 10000;
  int height = 10000;

  float *h_A = new float[width * height];
  float *h_B = new float[width * height];
  float *h_C = new float[width * height];

  FillMatrix(h_A, width, height, 1.0f);
  FillMatrix(h_B, width, height, 2.0f);

  float *A = NULL;
  float *B = NULL;
  float *C = NULL;

  size_t pA = 0;
  size_t pB = 0;
  size_t pC = 0;

  hipMallocPitch(&A, &pA, width, height);
  hipMallocPitch(&B, &pB, width, height);
  hipMallocPitch(&C, &pC, width, height);

  for (int row = 0; row < height; ++row) {
    float* rowA = (float*)((char*)A + row * pA);
    float* rowB = (float*)((char*)B + row * pB);
    hipMemcpy(rowA, h_A + row * width, width, hipMemcpyHostToDevice);
    hipMemcpy(rowB, h_B + row * width, width, hipMemcpyHostToDevice);
  }

  dim3 blockSize(256, 256);
  dim3 numBlocks((height + blockSize.x - 1) / blockSize.x, (width + blockSize.y - 1) / blockSize.y);

  KernelMatrixAdd<<<numBlocks, blockSize>>>(height, width, pA, pB, pC, A, B, C);
	hipDeviceSynchronize();

  for (int row = 0; row < height; ++row) {
    float* rowC = (float*)((char*)C + row * pC);
    hipMemcpy(h_C + row * width, rowC, width, hipMemcpyDeviceToHost);
  }

  for (int row = 0; row < height; ++row) {
    for(int col = 0; col < width; ++col) {
      assert(h_C[row * width + col] = 3.0f);
    }
  }

  hipFree(A);
  hipFree(B);
  hipFree(C);

  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}

