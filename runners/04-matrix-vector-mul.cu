#include <assert.h>
#include <MatrixVectorMul.cuh>

void FillMatrix(float* mat, int width, int height, float value) {
  for(int row = 0; row < height; ++row) {
    for(int col = 0; col < width; ++col) {
      mat[row * width + col] = value;
    }
  }
}

int main() {
  int width = 10000;
  int height = 10000;

  float *h_A = new float[width * height];
  float *h_vec = new float[width];
  float *h_res = new float[width];

  FillMatrix(h_A, width, height, 1.0f);
  FillMatrix(h_vec, width, 0, 2.0f);

  float *A = NULL;
  float *vec = NULL;
  float *res = NULL;

  hipMalloc(&A, width * height);
  hipMalloc(&vec, width);
  hipMalloc(&res, width);

  hipMemcpy(A, h_A, width * height, hipMemcpyHostToDevice);
  hipMemcpy(vec, h_vec, width, hipMemcpyHostToDevice);

  dim3 blockSize(256, 256);
  dim3 numBlocks((height + blockSize.x - 1) / blockSize.x, (width + blockSize.y - 1) / blockSize.y);

  MatrixVectorMul<<<numBlocks, blockSize>>>(height, width, A, vec, res);
	hipDeviceSynchronize();

  hipMemcpy(h_res, res, width, hipMemcpyDeviceToHost);

  for (int row = 0; row < width; ++row) {
    assert(h_res[row] = 2.0f * width);
  }

  hipFree(A);
  hipFree(vec);

  free(h_A);
  free(h_vec);

  return 0;
}

