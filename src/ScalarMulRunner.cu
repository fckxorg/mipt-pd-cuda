#include "hip/hip_runtime.h"
#include <ScalarMulRunner.cuh>


float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2, int blockSize) {
  return 0.0f;
}

float ScalarMulSumPlusReduction(int numElements, float* vector1, float* vector2, int blockSize) {
	int numBlocks = (numElements + blockSize - 1) / blockSize;
  //__shared__ float shared_data[];

  float *vec1_d = NULL;
  float *vec2_d = NULL;
  float *result_d = NULL;
  float *out_d = NULL;

  hipMalloc(&vec1_d, numElements * sizeof(float));
  hipMalloc(&vec2_d, numElements * sizeof(float));
  hipMalloc(&result_d, numBlocks * sizeof(float));
  hipMalloc(&out_d, sizeof(float));
  hipMemcpy(vec1_d, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vec2_d, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  ScalarMulBlock<<<numBlocks, blockSize>>>(numElements, vec1_d, vec2_d, result_d);
  numBlocks = numBlocks
  Reduce<<numBlocks, (numBlocks + blockSize - 1) / blockSize>>>(result_d, out_d);
  float result = 0;
  hipMemcpy(&result, out_d, sizeof(float), hipMemcpyDeviceToHost);
  return result;
}

