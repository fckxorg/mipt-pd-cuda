#include "hip/hip_runtime.h"
#include <ScalarMulRunner.cuh>
#include <ScalarMul.cuh>
#include <CommonKernels.cuh>


float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2, int blockSize) {
  return 0.0f;
}

float ScalarMulSumPlusReduction(int numElements, float* vector1, float* vector2, int blockSize) {
  const int numBlocks = (numElements + blockSize - 1) / blockSize;
  //__shared__ float shared_data[];

  float *vec1_d = NULL;
  float *vec2_d = NULL;
  float *result_d = NULL;
  float *out_d = NULL;

  hipMalloc(&vec1_d, numElements * sizeof(float));
  hipMalloc(&vec2_d, numElements * sizeof(float));
  hipMalloc(&result_d, numBlocks * sizeof(float));
  hipMalloc(&out_d, sizeof(float));
  hipMemcpy(vec1_d, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vec2_d, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  const int blockSizeReduce = (numBlocks + blockSize - 1) / blockSize;

  ScalarMulBlock<<<numBlocks, blockSize>>>(numElements, vec1_d, vec2_d, result_d);
  Reduce<<numBlocks, blockSizeReduce, numBlocks * sizeof(float)>>>(result_d, out_d);
  float result = 0;
  hipMemcpy(&result, out_d, sizeof(float), hipMemcpyDeviceToHost);
  return result;
}

