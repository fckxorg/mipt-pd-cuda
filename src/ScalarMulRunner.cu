#include "hip/hip_runtime.h"
#include <ScalarMulRunner.cuh>
#include <ScalarMul.cuh>
#include <CommonKernels.cuh>

__global__ void Reduce(float* in_data, float* out_data) {
    extern __shared__ float shared_data[];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = in_data[index];
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out_data[blockIdx.x] = shared_data[0];
    }
}



float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2, int blockSize) {
  return 0.0f;
}

float ScalarMulSumPlusReduction(int numElements, float* vector1, float* vector2, int blockSize) {
  const int numBlocks = (numElements + blockSize - 1) / blockSize;
  //__shared__ float shared_data[];

  float *vec1_d = NULL;
  float *vec2_d = NULL;
  float *result_d = NULL;
  float *out_d = NULL;

  hipMalloc(&vec1_d, numElements * sizeof(float));
  hipMalloc(&vec2_d, numElements * sizeof(float));
  hipMalloc(&result_d, numBlocks * sizeof(float));
  hipMalloc(&out_d, sizeof(float));
  hipMemcpy(vec1_d, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vec2_d, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  const int blockSizeReduce = (numBlocks + blockSize - 1) / blockSize;

  ScalarMulBlock<<<numBlocks, blockSize>>>(numElements, vec1_d, vec2_d, result_d);
  Reduce<<numBlocks, blockSizeReduce, numBlocks * sizeof(float)>>>(result_d, out_d);
  float result = 0;
  hipMemcpy(&result, out_d, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(vec1_d);
  hipFree(vec2_d);
  hipFree(result_d);
  hipFree(out_d);
  return result;
}

