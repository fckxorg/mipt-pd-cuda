#include "hip/hip_runtime.h"
#include <CommonKernels.cuh>

__global__ void Reduce(int* in_data, int* out_data) {
    extern __shared__ int shared_data[];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = in_data[index];
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out_data[blockIdx.x] = shared_data[0];
    }
}

__global__ void KernelMul(int numElements, float* x, float* y, float* result) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < numElements; i += stride) {
  result[i] = x[i] * y[i];
}
}

